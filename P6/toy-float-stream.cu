#include "hip/hip_runtime.h"
//=============================================================================
// FILE:   mytoy.cu
// AUTHORS: Raul Segura & Manuel Ujaldon (copyright 2014)
// Look for the string "MU" whenever Manuel suggests you to introduce changes
// Feel free to change some other parts of the code too (at your own risk)
//=============================================================================

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "io.h"

//=============================================================================
// CUDA functions.
//=============================================================================

//Error handler for CUDA functions.
void cudaErrorHandler(hipError_t error, const int LINE)
{
    if (error != hipSuccess) {
        fprintf(stdout, "ERROR(%d): %s\n", LINE, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

//-----------------------------------------------------------------------------
// Set the GPU device and get its properties.
void getDeviceProperties(const int devID, hipDeviceProp_t *deviceProp)
{
    // Set device.
    cudaErrorHandler(hipSetDevice(devID), __LINE__);

    // Get device properties.
    fprintf(stdout, "Leyendo propiedades del dispositivo %d...\n", devID);
    cudaErrorHandler(hipGetDeviceProperties(deviceProp, devID), __LINE__);

    fprintf(stdout, "GPU Device %d: \"%s\": capacidad de cómputo %d.%d.\n\n",
            devID, deviceProp->name, deviceProp->major, deviceProp->minor);
}

//=============================================================================
// IOHB functions (Input/Output Harwell-Boeing) adapted from the HB library
//=============================================================================

// Read the input matrix.
void readInputMatrix(const char *matrixFile, int *nrow, int *ncol, int *nnzero,
                     int **colptr, int **rowind, double **values)
{
    // Read the Harwell-Boeing format matrix file.
    fprintf(stdout, "Reading input matrix from %s...\n", matrixFile);
    readHB_newmat_double(matrixFile, nrow, ncol, nnzero,
                         colptr, rowind, values);

    fprintf(stdout, "Matrix in file %s is %d x %d ", matrixFile, *nrow, *ncol);
    fprintf(stdout, "with %d nonzero elements.\n\n", *nnzero);
}

//-----------------------------------------------------------------------------
// Write the output matrix.
void writeOutputMatrix(const char *matrixFile, int nrow, int ncol, int nnzero,
                       int *colptr, int *rowind, double *values)
{
    double *rhs = 0, *guess = 0, *exact = 0;
    char mxtype[] = "RUA";
    char ptrfmt[] = "(10I8)";
    char indfmt[] = "(10I8)";
    char valfmt[] = "(5E16.8)";
    char rhsfmt[] = "(5E16.8)";

    // Write the results of your computation into a file named "eureka",
    // which follows the Harwell-Boeing format.
    // POINT 1: Puedes cambiar el nombre "Eureka" si quieres comparar dos versiones de código diferentes.
    // O en caso de que quieras estar seguro de que algunas ejecuciones del mismo código producen exactamente el mismo resultado  (no race conditions occur when your
    // parallel strategy is deployed).
    //
    // Incluso podrías evitar llamar a esta función si la operación de salida es demasiado larga.
    fprintf(stdout, "Writing output matrix in %s...\n", matrixFile);
    writeHB_mat_double(matrixFile, nrow, ncol, nnzero, colptr, rowind, values,
                       0, rhs, guess, exact, matrixFile, "eureka", mxtype,
                       ptrfmt, indfmt, valfmt, rhsfmt, "FGN");

    fprintf(stdout, "Generated file %s successfully.\n\n", matrixFile);
}

//=============================================================================
// The CUDA Kernel.
//=============================================================================

// Cada hebra añade el elemento que le ha sido asignado a la matriz dispersa

// POINT 2: Cambia el tipo de dato a int, float or double
// You may want to change "float *dvalues" by "double *dvalues" in case
// you are curious to see how much GFLOPS drop when using double precision.
// Or even use "int dvalues" if you want to measure performance in integer ALUs.
// (see also hint MU4 below)
__global__ void kernelAdd(float *dvalues, int numOperations,
                          int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

// "numOperations" is the 2nd input parameter to our executable
    if (vi < nextColInd) {
        for (int j=0; j<numOperations; ++j) {
            // The operation performed on each nonzero of our sparse matrix:
            dvalues[vi] *=dvalues[vi]+dvalues[vi]*dvalues[vi]; // POINT 3: Choices you may try here:
        }                               // *= (for multiply), /= (for division),
    }                                   // or you may investigate some other :-)
}

//=============================================================================
// Main.
//=============================================================================

int main(int argc, char **argv)
{
	// =======================   Declaración de variables     ==================
	//=========================================================================
    // Variables.
    // CUDA.
    hipDeviceProp_t deviceProp;
    hipStream_t *stream;
    hipEvent_t start, stop;

    // Matrix.
    // Harwell-Boeing format.
    int nrow, ncol, nnzero;
    // Compressed Sparse Column format.
    int *colptr, *rowind;
    float *values;  // POINT 4: Puedes usar int para medir el rendimeinto en operaciones en punto fijo
    // o double para doble precisión
    double *values64; //

    // To measure time elapsed and performance achieved
    float msecMemHst, msecMemDvc, msecCompStr, msecCompKrn;
    float numOperationsPerValue, numFloatingPointOperations, opIntensity;
    double flops, gigaFlops;

    // Misc.
    int devID;
    int *blocks;
    int *threads;
    float *dvalues;  // POINT 5: This declaration is binded to hints MU2 and MU4

    // ======================= Comprobación de parámetros de entrada ==================
    //=========================================================================
    // Check command line arguments.
    if (argc < 5) {
        fprintf(stderr, "ERROR: Número equivocado de argumentos: %d\n", argc - 1);
        fprintf(stderr, "Use: ./mytoy <deviceID> <numOperationsPer");
        fprintf(stderr, "Value> <inputMatrixFile> <outputMatrixFile>\n");
        exit(EXIT_FAILURE);
    }

     //-------------------------------------------------------------------------
    // This part is just to restrict the execution to device (GPU) 0 or 1
    devID = atoi(argv[1]);
    if ((devID != 0) && (devID != 1)) {
        fprintf(stderr, "ERROR: El primero parámetro es   %s.\n", argv[1]);
        fprintf(stderr, "Tiene que ser 0 para seleccionar el dispositivo GPU en el que vamos a ejecutar.");
        exit(EXIT_FAILURE);
    }

    numOperationsPerValue = atoi(argv[2]);
    if (numOperationsPerValue <= 0) {
        fprintf(stderr, "ERROR: El segundo parámetro es incorrecto: %s.\n", argv[2]);
        fprintf(stderr, "Representa el número de operaciones por valor y debe ser mayor que 0 ");
        exit(EXIT_FAILURE);
    }

    // ======================= Lectura de las características de la tarjeta ==================
    //=========================================================================
    // Get properties of the chosen device.
    getDeviceProperties(devID, &deviceProp);

    // =================== Creación de eventos para monitorizar el tiempo ========
    //-------------------------------------------------------------------------
    // Create CUDA events for timing.
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //======================  Lectura de la matriz de entrada ===================================================
    // Lee la matriz de entrada.
    readInputMatrix(argv[3], &nrow, &ncol, &nnzero,
                    &colptr, &rowind, &values64);
    fprintf(stderr,"Tamaño de la matriz, nrow=%d, ncol=%d\n",nrow,ncol);

    // ======================= Reserva de memoria ==================
    //  POINT 6: Aquí hay que especificar el tipo de dato que puede ser float, double o int (ver Punto 2, punto 4 y punto 5)
    values = (float*)malloc(nnzero * sizeof(float));
    for (int i=0; i<nnzero; ++i) {
    //  POINT 7: No olvides cambiar el casting según la declaración del punto 2, 4, 5 y 6
        values[i] = (float)values64[i];
    }

    // ======================= Valores para calcular los bloques y el número de hebras por bloque  ==================

    // Maximum number of threads per block and warp size.
    int maxThreadsPerBlock = 1024;
    const int warpSize = 32;  // Esto no se puede cambiar, no es optativo.

    // ======================= Calculo del grid de hebras ==================

    // Calcular el número de bloques y de hebras que necesitamos para cada columna
    // POINT 8: Aquí tienes que establecer el tamaño del grid para sacarle todo el paralelismo que puedas al lanzamiento del kernel (Ver punto 11)
    blocks = (int*)malloc(ncol * sizeof(float));
    threads = (int*)malloc(ncol * sizeof(float));

    // ======================= Cálculo de los diferentes grids que vamos a lanzar  ==================

    for (int i=0; i<ncol; ++i) {
        fprintf(stderr,"Para i=%d, (((colptr[%d](%d) - colptr[%d](%d))/%d)+1)*%d",i, i+1,colptr[i+1],i,colptr[i],warpSize,warpSize);

        threads[i] = (((colptr[i+1] - colptr[i]) / warpSize) + 1) * warpSize;
        fprintf(stderr,"->>>> threads[%d]=%d",i,threads[i]);

        if (threads[i] <= maxThreadsPerBlock) {
            blocks[i] = 1;
        } else {
            blocks[i] = threads[i] / maxThreadsPerBlock;
            if (threads[i] % maxThreadsPerBlock > 0) {blocks[i]++;}
            threads[i] = maxThreadsPerBlock;
            fprintf(stderr,"->>>> threads[%d]=%d",i,threads[i]);

        }
        fprintf(stderr,"->>>> Blocks[%d]=%d\n",i,blocks[i]);
    }


    //=========================  Ejecución de los grids ===================================
    // Ejecución
    fprintf(stdout, "Running mytoy.\n");

    //-------------------------------------------------------------------------
    // Copy matrix values from host memory to device memory.

    //PUNTO  9: Hay que adecuar el tipo de dato a float, int o double (Ver puntos 2, 4, 5, 6 y 7)
    int valuesSize = nnzero * sizeof(float);

    cudaErrorHandler(hipEventRecord(start, NULL), __LINE__);

    //fprintf(stdout, "Reservando %d bytes en la memoria del ", valuesSize);
    //fprintf(stdout, "dispositivo para los valores del array ...\n");
    cudaErrorHandler(hipMalloc((void**)&dvalues, valuesSize), __LINE__);

    //fprintf(stdout, "Copiando datos desde la memoria del host hasta la memoria del dispositivo...\n");
    cudaErrorHandler(hipMemcpy(dvalues, values, valuesSize,
                                hipMemcpyHostToDevice), __LINE__);

    cudaErrorHandler(hipEventRecord(stop, NULL), __LINE__); // Registra el momento del evento de finalización de la copia de la memoria
    cudaErrorHandler(hipEventSynchronize(stop), __LINE__);
    cudaErrorHandler(hipEventElapsedTime(&msecMemHst, start, stop), __LINE__); // Calcula el tiempo transcurridos con una precisión de 0.5 microsegundos

    //-------------------------------------------------------------------------
    // Create streams.
    cudaErrorHandler(hipEventRecord(start, NULL), __LINE__); // Comienza el siguiente tramo de código

    // 	PUNTO 10: Si crees que un sólo stream es mejor para toda la matriz,
    // sólo tienes que reemplazar la siguiente sentencia y el bucle por la siguiente línea
     // cudaErrorHandler(hipStreamCreate(&stream), __LINE__);

    stream = (hipStream_t*)malloc(ncol * sizeof(hipStream_t));
    for (int i=0; i<ncol; ++i) {
        cudaErrorHandler(hipStreamCreate(&stream[i]), __LINE__);
    }
    //fprintf(stdout, "Stream(s) Creado correctamente.\n");

    cudaErrorHandler(hipEventRecord(stop, NULL), __LINE__); // Registra la finalización del evento
    cudaErrorHandler(hipEventSynchronize(stop), __LINE__); // Sincroniza
    cudaErrorHandler(hipEventElapsedTime(&msecCompStr, start, stop),__LINE__); // Calcula el tiempo

    //-------------------------------------------------------------------------
    // Launch streams.
    cudaErrorHandler(hipEventRecord(start, NULL), __LINE__); // Comienza el lanzamiento

    //fprintf(stdout, "Lanzando un stream por columna...\n");
    for (int i=0; i<ncol; ++i) { // PUNTO 11: La forma en la que se despliega el paralelismo está aquí.
    	// Reemplaza stream[i] por stream en la siguiente línea si has hecho el cambio del punto 9
        kernelAdd<<<blocks[i], threads[i], 0, stream>>>(dvalues, numOperationsPerValue, colptr[i], colptr[i+1]);
    }
    //fprintf(stdout, "Ejecutando los streams...\n");

    cudaErrorHandler(hipEventRecord(stop, NULL), __LINE__);
    cudaErrorHandler(hipEventSynchronize(stop), __LINE__);
    cudaErrorHandler(hipEventElapsedTime(&msecCompKrn, start, stop),__LINE__);

    cudaErrorHandler(hipDeviceSynchronize(), __LINE__);
    fprintf(stdout, "Streams executed successfully.\n");

    //-------------------------------------------------------------------------
    // Copiar los resultados de vuelta a la CPU.
    cudaErrorHandler(hipEventRecord(start, NULL), __LINE__);

    fprintf(stdout, "Copiando los valores de vuelta desde la... ");
    fprintf(stdout, "memoria del dispositivo hasta la memoria del host...\n\n");
    cudaErrorHandler(hipMemcpy(values, dvalues, valuesSize,
                                hipMemcpyDeviceToHost), __LINE__);

    cudaErrorHandler(hipEventRecord(stop, NULL), __LINE__);
    cudaErrorHandler(hipEventSynchronize(stop), __LINE__);
    cudaErrorHandler(hipEventElapsedTime(&msecMemDvc, start, stop), __LINE__);

    //=======================Escribir matriz de salida ======================
    // Escribir la matriz de salida
    for (int i=0; i<nnzero; ++i) {
        values64[i] = (double)values[i];
    }

    writeOutputMatrix(argv[4], nrow, ncol, nnzero,
                      colptr, rowind, values64);


    // ======================= Calculo de rendimiento ==================

    // Imprimiendo tiempos y porcentages.
    float msecMem = msecMemHst + msecMemDvc;
    float msecComp = msecCompStr + msecCompKrn;
    fprintf(stdout, "Tiempo de acceso a la memoria de la GPU: %.4f ms.\n\n", msecMem);

    fprintf(stdout, "Creación de streams en la GPU:  %.4f ms.\n", msecCompStr);
    fprintf(stdout, "Tiempo de ejecución del kernel: %.4f ms.\n", msecCompKrn);
    fprintf(stdout, "Tiempo de computación en GPU:   %.4f ms.\n\n", msecComp);

    //PUNTO 12:	Cambia float, int or double según el punto 2, 4, 5, 6, 7 y 8
    opIntensity = numOperationsPerValue / sizeof(float);
    fprintf(stdout, "Operaciones en punto flotante por byte: %.4f FLOP/byte.\n", opIntensity);

    numFloatingPointOperations = nnzero * numOperationsPerValue;
    flops = numFloatingPointOperations / (msecComp / 1000.0f);
    gigaFlops = flops * 1.0e-9f;
    fprintf(stdout, "Rendimiento: %.4f GFLOP/s.\n\n", gigaFlops);

    //=========================================================================
    // Free host memory.
    free(colptr); free(rowind); free(values);
    free(blocks); free(threads);

    // liberación.
    cudaErrorHandler(hipDeviceReset(), __LINE__);

    return EXIT_SUCCESS;
}
